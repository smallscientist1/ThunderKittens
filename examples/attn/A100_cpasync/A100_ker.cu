#include "hip/hip_runtime.h"
#include "../../../src/kittens.cuh"

#include <cuda/barrier>
#include <hip/hip_cooperative_groups.h>

 // this kernel is more of an example kernel to show some TK programming models, rather than a kernel we think you should put into production, though it is pretty fast!

#define NUM_WORKERS 8 // This kernel uses 8 workers in parallel per block, to help issue instructions more quickly.

using namespace kittens; // this kernel only handles headdim=64 for simplicity. Also n should be a multiple of 256 here.
__global__ void attend_ker64(int n, const bf16* __restrict__ __q__, const bf16* __restrict__ __k__, const bf16* __restrict__ __v__, bf16* __o__) {

    auto warpid        = kittens::warpid();
    // auto block_start   = blockIdx.x*(n*64);
    auto block_start_q = blockIdx.y * (n * 64) + blockIdx.x * (NUM_WORKERS * kittens::TILE_DIM * 64);
    auto block_start_kv = blockIdx.y * (n * 64);
    // const bf16 *_q = __q__ + block_start, *_k = __k__ + block_start, *_v = __v__ + block_start;
    //       bf16 *_o = __o__ + block_start;
    const bf16 *_q = __q__ + block_start_q, *_k = __k__ + block_start_kv, *_v = __v__ + block_start_kv;
          bf16 *_o = __o__ + block_start_q;

    extern __shared__ alignment_dummy __shm[]; // this is the CUDA shared memory
    shared_allocator al((int*)&__shm[0]);
    
    // K and V live in shared memory -- this is about all that will fit.
    st_bf_1x4<ducks::st_layout::swizzle> (&k_smem)[2][NUM_WORKERS] = al.allocate<st_bf_1x4<ducks::st_layout::swizzle>, 2, NUM_WORKERS>();
    st_bf_1x4<ducks::st_layout::swizzle> (&v_smem)[2][NUM_WORKERS] = al.allocate<st_bf_1x4<ducks::st_layout::swizzle>, 2, NUM_WORKERS>();
    using barrier = cuda::barrier<cuda::thread_scope_block>;
    __shared__ barrier bar;
    // barrier bar;
    if (threadIdx.x == 0) {
        init(&bar, NUM_WORKERS*32);
    }
    __syncthreads();

    int tic = 0, toc = 0;

    // Initialize all of the register tiles.
    rt_bf_1x4<> q_reg, k_reg, v_reg; // v_reg need to be swapped into col_l
    rt_fl_1x1<> att_block;
    rt_bf_1x1<> att_block_mma;
    rt_fl_1x4<> o_reg;
    rt_fl_1x1<>::col_vec max_vec_last, max_vec; // these are column vectors for the attention block
    rt_fl_1x1<>::col_vec norm_vec_last, norm_vec; // these are column vectors for the attention block
    
    // int qo_blocks = n / (q_reg.rows*NUM_WORKERS), kv_blocks = n / (q_reg.rows*NUM_WORKERS);
    int kv_blocks = n / (q_reg.rows*NUM_WORKERS);

    // for(auto q_blk = 0; q_blk < qo_blocks; q_blk++) {

        // each warp loads its own Q tile of 16x64, and then multiplies by 1/sqrt(d)
        // load(q_reg, _q + (q_blk*NUM_WORKERS + warpid)*q_reg.num_elements, q_reg.cols);
        load(q_reg, _q + (warpid)*q_reg.num_elements, q_reg.cols);
        mul(q_reg, q_reg, __float2bfloat16(0.125f)); // temperature adjustment

        // zero flash attention L, M, and O registers.
        neg_infty(max_vec); // zero registers for the Q chunk
        zero(norm_vec);
        zero(o_reg);

        load_async(k_smem[tic][warpid], _k + (0*NUM_WORKERS + warpid)*q_reg.num_elements, q_reg.cols, bar);
        load_async(v_smem[tic][warpid], _v + (0*NUM_WORKERS + warpid)*q_reg.num_elements, q_reg.cols, bar);
        tic ^= 1;

        // iterate over k, v for these q's that have been loaded
        for(auto kv_idx = 0; kv_idx < kv_blocks; kv_idx++, tic ^= 1, toc ^= 1) {

            // each warp loads its own chunk of k, v into shared memory
            // load(v_smem[warpid], _v + (kv_idx*NUM_WORKERS + warpid)*q_reg.num_elements, q_reg.cols);
            // load(k_smem[warpid], _k + (kv_idx*NUM_WORKERS + warpid)*q_reg.num_elements, q_reg.cols);
            // __syncthreads(); // we need to make sure all memory is loaded before we can begin the compute phase

            bar.arrive_and_wait();
            __syncthreads();
            if (kv_idx < kv_blocks-1) {
                load_async(v_smem[tic][warpid], _v + ((kv_idx+1)*NUM_WORKERS + warpid)*q_reg.num_elements, q_reg.cols, bar);
                load_async(k_smem[tic][warpid], _k + ((kv_idx+1)*NUM_WORKERS + warpid)*q_reg.num_elements, q_reg.cols, bar);
            }

            // bar.arrive_and_wait(); // Correct
            // bar.arrive(); // Incorrect
            // bar.wait(bar.arrive()); // Correct, arrive returns the constructed arrival_token object.
            // __syncthreads(); // we need to make sure all memory is loaded before we can begin the compute phase

            // now each warp goes through all of the subtiles, loads them, and then does the flash attention internal alg.
            for(int subtile = 0; subtile < NUM_WORKERS; subtile++) {

                load(k_reg, k_smem[toc][subtile]); // load k from shared into registers

                zero(att_block); // zero 16x16 attention tile
                mma_ABt(att_block, q_reg, k_reg, att_block); // Q@K.T

                copy(norm_vec_last, norm_vec);
                copy(max_vec_last,  max_vec);

                row_max(max_vec, att_block, max_vec); // accumulate onto the max_vec
                sub_row(att_block, att_block, max_vec); // subtract max from attention -- now all <=0
                exp(att_block, att_block); // exponentiate the block in-place.

                sub(max_vec_last, max_vec_last, max_vec); // subtract new max from old max to find the new normalization.
                exp(max_vec_last, max_vec_last); // exponentiate this vector -- this is what we need to normalize by.
                mul(norm_vec, norm_vec, max_vec_last); // and the norm vec is now normalized.

                row_sum(norm_vec, att_block, norm_vec); // accumulate the new attention block onto the now-rescaled norm_vec
                div_row(att_block, att_block, norm_vec); // now the attention block is correctly normalized

                mul(norm_vec_last, norm_vec_last, max_vec_last); // normalize the previous norm vec according to the new max
                div(norm_vec_last, norm_vec_last, norm_vec); // normalize the previous norm vec according to the new norm

                copy(att_block_mma, att_block); // convert to bf16 for mma_AB

                load(v_reg, v_smem[toc][subtile]); // load v from shared into registers.
                rt_bf_1x4<ducks::rt_layout::col> &v_reg_col = swap_layout_inplace(v_reg); // this is a reference and the call has invalidated v_reg

                mul_row(o_reg, o_reg, norm_vec_last); // normalize o_reg in advance of mma_AB'ing onto it
                mma_AB(o_reg, att_block_mma, v_reg_col, o_reg); // mfma onto o_reg with the local attention@V matmul.
            }
            // __syncthreads(); // we need to make sure all warps are done before we can start loading the next kv chunk
        }

        // store(_o + (q_blk*NUM_WORKERS + warpid)*q_reg.num_elements, o_reg, q_reg.cols); // write out o. compiler has an issue with register usage if d is made constexpr q_reg.rows :/
        store(_o + (warpid)*q_reg.num_elements, o_reg, q_reg.cols); // write out o. compiler has an issue with register usage if d is made constexpr q_reg.rows :/
    // }
}

#include "harness.impl"
