#include "hip/hip_runtime.h"
// #define TORCH_COMPILE // defined by default for PyTorch bindings - to use cpp harness, comment this out

#ifdef TORCH_COMPILE
#include "../src/kittens.cuh"
#else
#include "../../../src/kittens.cuh"
#endif

#include <cuda/pipeline>
#include <hip/hip_cooperative_groups.h>

#define NUM_WORKERS (8)
// #define NUM_WARPGROUPS (NUM_WORKERS/(kittens::WARPGROUP_WARPS))
#define NUM_WORKERS_KV (4)

using namespace kittens;

template<ducks::rt::row_layout RT>
__device__ static inline void wg_make_causal(RT &dst, const RT &src, const typename base_types::packing<typename RT::dtype>::unpacked_type &val=0) {
    const typename RT::dtype packed_val = base_types::packing<typename RT::dtype>::pack(val);
    #pragma unroll
    for(int i = 0; i < dst.height; i++) {
        #pragma unroll
        for(int j = 0; j < dst.width; j++) {

            if(j < ((warpid() % kittens::WARPGROUP_WARPS) * dst.height) + i) { // below the diagonal, copy
                #pragma unroll
                for(int k = 0; k < dst.packed_per_tile; k++) {
                    dst.tiles[i][j].data[k] = src.tiles[i][j].data[k];
                }
            }
            else if(j > ((warpid() % kittens::WARPGROUP_WARPS) * dst.height) + i) { // above the diagonal, zero
                #pragma unroll
                for(int k = 0; k < dst.packed_per_tile; k++) {
                    dst.tiles[i][j].data[k] = packed_val;
                }
            }
            else { // on the diagonal, interesting!
                constexpr uint32_t MASK_X = 0xFF773311, MASK_Y = 0xF7733110; // magic numbers for on-diagonal core matrices
                dst.tiles[i][j].data[1] = src.tiles[i][j].data[1]; // below diagonal, copy
                dst.tiles[i][j].data[2] = packed_val; // above diagonal, zero
                if((MASK_X >> laneid()) & 1) {
                    dst.tiles[i][j].data[0].x = src.tiles[i][j].data[0].x;
                    dst.tiles[i][j].data[3].x = src.tiles[i][j].data[3].x;
                }
                else {
                    dst.tiles[i][j].data[0].x = val;
                    dst.tiles[i][j].data[3].x = val;
                }
                if((MASK_Y >> laneid()) & 1) {
                    dst.tiles[i][j].data[0].y = src.tiles[i][j].data[0].y;
                    dst.tiles[i][j].data[3].y = src.tiles[i][j].data[3].y;
                }
                else {
                    dst.tiles[i][j].data[0].y = val;
                    dst.tiles[i][j].data[3].y = val;
                }
            }
        }
    }
}

// warp 间一定是 warp*1 排布
template<ducks::rt::row_layout RT>
__device__ static inline void warp_make_causal(RT &dst, const RT &src, const typename base_types::packing<typename RT::dtype>::unpacked_type &val=0) {
    const typename RT::dtype packed_val = base_types::packing<typename RT::dtype>::pack(val);
    #pragma unroll
    for(int i = 0; i < dst.height; i++) {
        #pragma unroll
        for(int j = 0; j < dst.width; j++) {

            if(j < ((warpid() ) * dst.height) + i) { // below the diagonal, copy
                #pragma unroll
                for(int k = 0; k < dst.packed_per_tile; k++) {
                    dst.tiles[i][j].data[k] = src.tiles[i][j].data[k];
                }
            }
            else if(j > ((warpid() ) * dst.height) + i) { // above the diagonal, zero
                #pragma unroll
                for(int k = 0; k < dst.packed_per_tile; k++) {
                    dst.tiles[i][j].data[k] = packed_val;
                }
            }
            else { // on the diagonal, interesting!
                constexpr uint32_t MASK_X = 0xFF773311, MASK_Y = 0xF7733110; // magic numbers for on-diagonal core matrices
                dst.tiles[i][j].data[1] = src.tiles[i][j].data[1]; // below diagonal, copy
                dst.tiles[i][j].data[2] = packed_val; // above diagonal, zero
                if((MASK_X >> laneid()) & 1) {
                    dst.tiles[i][j].data[0].x = src.tiles[i][j].data[0].x;
                    dst.tiles[i][j].data[3].x = src.tiles[i][j].data[3].x;
                }
                else {
                    dst.tiles[i][j].data[0].x = val;
                    dst.tiles[i][j].data[3].x = val;
                }
                if((MASK_Y >> laneid()) & 1) {
                    dst.tiles[i][j].data[0].y = src.tiles[i][j].data[0].y;
                    dst.tiles[i][j].data[3].y = src.tiles[i][j].data[3].y;
                }
                else {
                    dst.tiles[i][j].data[0].y = val;
                    dst.tiles[i][j].data[3].y = val;
                }
            }
        }
    }
}

using layout_q = kittens::ducks::st_layout::swizzle; 
using layout_k = kittens::ducks::st_layout::swizzle; 
using layout_v = kittens::ducks::st_layout::swizzle; 
using layout_o = kittens::ducks::st_layout::swizzle;

// shared tile 128, 128（must be square for causal）
constexpr int qo_height = 8; 
constexpr int kv_height = 8; // not used

// NUMWORKERS * 16 = qo_height(shared tile)
template<int D>
__global__  __launch_bounds__((NUM_WORKERS)*kittens::WARP_THREADS, 2)
void fwd_attend_ker_dim(int N, const bf16* __restrict__ __q__, const bf16* __restrict__ __k__, const bf16* __restrict__ __v__, bf16* __o__) {
    extern __shared__ int __shm[]; // this is the CUDA shared memory
    shared_allocator al((int*)&__shm[0]);

    auto block_start = blockIdx.x * (N * D);
    const bf16 *_q = __q__ + block_start, *_k = __k__ + block_start, *_v = __v__ + block_start;
          bf16 *_o = __o__ + block_start;

    // st_bf<qo_height, D/kittens::TILE_DIM, layout_q>          (&q_smem)   [NUM_WARPGROUPS] = al.allocate<st_bf<qo_height, D/kittens::TILE_DIM, layout_q>,          NUM_WARPGROUPS>();
    st_bf<1, D/kittens::TILE_DIM, layout_k>          (&k_smem)[2][NUM_WORKERS_KV] = al.allocate<st_bf<1, D/kittens::TILE_DIM, layout_k>, 2,       NUM_WORKERS_KV>();
    st_bf<1, D/kittens::TILE_DIM, layout_v>          (&v_smem)[2][NUM_WORKERS_KV] = al.allocate<st_bf<1, D/kittens::TILE_DIM, layout_v>, 2,       NUM_WORKERS_KV>();

    int tic = 0, toc = 1;
 
    rt_bf<1, D/kittens::TILE_DIM> q_reg, k_reg, v_reg;
    // rt_fl<1, kv_height> att_block;
    // rt_bf<1, kv_height> att_block_mma;
    rt_fl<1, 1> att_block;
    rt_bf<1, 1> att_block_mma;
    rt_fl<1, D/kittens::TILE_DIM> o_prev;
    rt_fl<1, kv_height>::col_vec max_vec_last,  max_vec;
    rt_fl<1, kv_height>::col_vec norm_vec_last, norm_vec;

    int warpid      = kittens::warpid();
    // int warpgroupid = warpid/kittens::WARPGROUP_WARPS;

    int qo_index    = (blockIdx.x * NUM_WORKERS) + warpid;

    int kv_blocks = N / (NUM_WORKERS_KV*k_smem[0][0].rows);

    // __shared__ uint64_t qsmem_barrier, kvsmem_barrier;//, vsmem_barrier;

    // int q_phasebit = 0;
    // int kv_phasebit = 0;

    // if (threadIdx.x == 0) {
    //     tma::init_barrier<st_bf<qo_height, D/kittens::TILE_DIM, layout_q>, NUM_WARPGROUPS>(qsmem_barrier, 1);
    //     tma::init_barrier<st_bf<kv_height, D/kittens::TILE_DIM, layout_k>, NUM_WORKERS_KV*2>(kvsmem_barrier, 1); 
    // }

    // if (warpid == 0) {
    //     for (int wg = 0; wg < NUM_WORKERS/kittens::WARPGROUP_WARPS; wg++) { // load q
    //         int tile_idx = (blockIdx.y * NUM_WARPGROUPS * gridDim.x) + (blockIdx.x * NUM_WARPGROUPS) + wg;
    //         tma::load_async((q_smem[wg]), tma_q, qsmem_barrier, tile_idx); 
    //     }
    //     for (int w = 0; w < NUM_WORKERS_KV; w++) { // load k, v      
    //         int tile_idx = (blockIdx.y * NUM_WORKERS_KV * kv_blocks) + (0 * NUM_WORKERS_KV) + w; 
    //         tma::load_async((k_smem[tic][w]), tma_k, kvsmem_barrier, tile_idx); 
    //         tma::load_async((v_smem[tic][w]), tma_v, kvsmem_barrier, tile_idx); 
    //     }
    // }

    load(q_reg, _q + (warpid)*q_reg.num_elements, q_reg.cols);


    neg_infty(max_vec); // zero registers for the Q chunk
    zero(norm_vec);
    zero(o_prev);
    // __syncthreads();

    // tma::arrive_and_wait(qsmem_barrier, q_phasebit);
    // q_phasebit ^= 1;

    // if constexpr (D == 64) { warpgroup::mul(q_smem[warpgroupid], q_smem[warpgroupid], __float2bfloat16(0.125f)); }
    // else { warpgroup::mul(q_smem[warpgroupid], q_smem[warpgroupid], __float2bfloat16(0.08838834764f)); }
    if constexpr (D == 64) { mul(q_reg, q_reg, __float2bfloat16(0.125f)); }
    else { mul(q_reg, q_reg, __float2bfloat16(0.08838834764f)); }


    for (auto kv_idx = 0; kv_idx <= qo_index/NUM_WORKERS; kv_idx++, tic ^= 1, toc ^= 1) {
        // tma::arrive_and_wait(kvsmem_barrier, kv_phasebit);
        // load from global, global no swizzle; thread mapping default
        load(k_smem[tic][warpid], _k + (kv_idx*NUM_WORKERS+warpid)*k_reg.num_elements, k_reg.cols);
        load(v_smem[tic][warpid], _v + (kv_idx*NUM_WORKERS+warpid)*v_reg.num_elements, v_reg.cols);
        // kv_phasebit ^= 1;

        __syncthreads();
        // if (warpid == 0) {
        //     if (kv_idx + 1 < kv_blocks) {
        //         tma::set_bytes(kvsmem_barrier, 2 * NUM_WORKERS_KV * k_smem[0][0].num_elements * sizeof(bf16));
                
        //         for (int w = 0; w < NUM_WORKERS_KV; w++) {        
        //             int tile_idx = (blockIdx.y * NUM_WORKERS_KV * kv_blocks) + ((kv_idx + 1) * NUM_WORKERS_KV) + w; 
        //             tma::load_async((k_smem[toc][w]), tma_k, kvsmem_barrier, tile_idx); 
        //             tma::load_async((v_smem[toc][w]), tma_v, kvsmem_barrier, tile_idx);
        //         }
        //     }
        // }

        // warpgroup::mma_fence(att_block);
        // warpgroup::mm_ABt(att_block, q_smem[warpgroupid], k_smem[tic][0]);
        // warpgroup::mma_commit_group();
        for(int subtile = 0; subtile < NUM_WORKERS_KV; subtile++) { // 1
            load(k_reg, k_smem[tic][subtile]);
            zero(att_block);
            mma_ABt(att_block, q_reg, k_reg, att_block);

            copy(norm_vec_last, norm_vec);
            copy(max_vec_last,  max_vec);

            if (kv_idx == qo_index/NUM_WORKERS) {
                warp_make_causal(att_block, att_block, -INFINITY); 
            }

            row_max(max_vec, att_block, max_vec); // accumulate onto the max_vec
            sub_row(att_block, att_block, max_vec);
            exp(att_block, att_block);

            sub(max_vec_last, max_vec_last, max_vec);
            exp(max_vec_last, max_vec_last);
            mul(norm_vec, norm_vec, max_vec_last);

            row_sum(norm_vec, att_block, norm_vec); // accumulate onto the norm_vec
            div_row(att_block, att_block, norm_vec);

            mul(norm_vec_last, norm_vec_last, max_vec_last);
            div(norm_vec_last, norm_vec_last, norm_vec);

            copy(att_block_mma, att_block); // convert to bf16 for mma

            load(v_reg, v_smem[tic][subtile]);
            // different smem layout for mma_AB vs mma_ABt to avoid load smem->reg smem bankconflicts
            rt_bf<1, D/kittens::TILE_DIM, ducks::rt_layout::col> &v_reg_col = swap_layout_inplace(v_reg); // this is a reference and the call has invalidated v_reg
            

            mul_row(o_prev, o_prev, norm_vec_last); // normalize o_prev in advance of mma'ing onto it
            mma_AB(o_prev, att_block_mma, v_reg_col, o_prev); // mfma onto o_prev with the local attention@V matmul.
        }
        // warpgroup::mma_fence(o_prev);
        // warpgroup::mma_AB(o_prev, att_block_mma, v_smem[tic][0]);
        // warpgroup::mma_commit_group();
    }

    // auto (*o_smem) = reinterpret_cast<st_bf<qo_height, D/kittens::TILE_DIM, layout_o>(*)>(q_smem); // reuse q memory
    // warpgroup::store(o_smem[warpgroupid], o_prev); 
    // __syncthreads();
    
    // if (warpid % 4 == 0) { // store o
    //     int tile_idx = (blockIdx.y * NUM_WARPGROUPS * gridDim.x) + (blockIdx.x * NUM_WARPGROUPS) + warpgroupid;
    //     tma::store_async(tma_o, (o_smem[warpgroupid]), tile_idx); 
    //     tma::store_commit_group(); 
    // }

    // tma::store_async_wait();
    store(_o + (warpid)*o_prev.num_elements, o_prev, o_prev.cols);
}

#ifdef TORCH_COMPILE
#include "src/common/pyutils/torch_helpers.cuh"
#include <iostream>

void attention_forward_causal(torch::Tensor q, torch::Tensor k, torch::Tensor v, torch::Tensor o) {

    CHECK_INPUT(q);
    CHECK_INPUT(k);
    CHECK_INPUT(v);
    CHECK_INPUT(o);

    auto batch   = q.size(0);
    auto heads   = q.size(1);
    auto N       = q.size(2);
    auto D       = q.size(3);

    auto threads = NUM_WORKERS * kittens::WARP_THREADS;

    TORCH_CHECK(q.scalar_type() == c10::ScalarType::BFloat16, "q must be bf16");
    TORCH_CHECK(k.scalar_type() == c10::ScalarType::BFloat16, "k must be bf16");
    TORCH_CHECK(v.scalar_type() == c10::ScalarType::BFloat16, "v must be bf16");
    TORCH_CHECK(o.scalar_type() == c10::ScalarType::BFloat16, "o must be bf16");

    // make sure sequence length is multiple of 128 for now
    TORCH_CHECK(N % (NUM_WORKERS * kittens::TILE_DIM) == 0, "Please pad sequence length to be multiple of 128");

    // make sure D = 64 or 128
    TORCH_CHECK(D == 64 | D == 128, "Currently, only D = 64 or 128 is supported");

    // convert to bf16
    c10::BFloat16 *q_ptr = q.data_ptr<c10::BFloat16>();
    c10::BFloat16 *k_ptr = k.data_ptr<c10::BFloat16>();
    c10::BFloat16 *v_ptr = v.data_ptr<c10::BFloat16>();
    c10::BFloat16 *o_ptr = o.data_ptr<c10::BFloat16>();

    const bf16* q_bf = reinterpret_cast<const bf16*>(q_ptr);
    const bf16* k_bf = reinterpret_cast<const bf16*>(k_ptr);
    const bf16* v_bf = reinterpret_cast<const bf16*>(v_ptr);
    bf16* o_bf = reinterpret_cast<bf16*>(o_ptr);

    if (D == 64) {
        CUtensorMap* tma_q_d = tma::allocate_and_create_tensor_map<kittens::st_bf<qo_height, 4, layout_q>>(q_bf, (batch*heads*N)/(qo_height * 16));
        CUtensorMap* tma_k_d = tma::allocate_and_create_tensor_map<kittens::st_bf<kv_height, 4, layout_k>>(k_bf, (batch*heads*N)/(kv_height * 16));
        CUtensorMap* tma_v_d = tma::allocate_and_create_tensor_map<kittens::st_bf<kv_height, 4, layout_v>>(v_bf, (batch*heads*N)/(kv_height * 16));
        CUtensorMap* tma_o_d = tma::allocate_and_create_tensor_map<kittens::st_bf<qo_height, 4, layout_o>>(o_bf, (batch*heads*N)/(qo_height * 16));

        unsigned long mem_size = 112000;
        hipFuncSetAttribute(reinterpret_cast<const void*>(fwd_attend_ker_dim<64>), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);

        dim3 grid(N/(NUM_WORKERS*kittens::TILE_DIM), batch*heads, 1);

        fwd_attend_ker_dim<64><<<grid, threads, mem_size>>>(N, tma_q_d, tma_k_d, tma_v_d, tma_o_d);
    }
    else {
        CUtensorMap* tma_q_d = tma::allocate_and_create_tensor_map<kittens::st_bf<qo_height, 8, layout_q>>(q_bf, (batch*heads*N)/(qo_height * 16));
        CUtensorMap* tma_k_d = tma::allocate_and_create_tensor_map<kittens::st_bf<kv_height, 8, layout_k>>(k_bf, (batch*heads*N)/(kv_height * 16));
        CUtensorMap* tma_v_d = tma::allocate_and_create_tensor_map<kittens::st_bf<kv_height, 8, layout_v>>(v_bf, (batch*heads*N)/(kv_height * 16));
        CUtensorMap* tma_o_d = tma::allocate_and_create_tensor_map<kittens::st_bf<qo_height, 8, layout_o>>(o_bf, (batch*heads*N)/(qo_height * 16));

        unsigned long mem_size = 112000;
        hipFuncSetAttribute(reinterpret_cast<const void*>(fwd_attend_ker_dim<128>), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);

        dim3 grid(N/(NUM_WORKERS*kittens::TILE_DIM), batch*heads, 1);

        fwd_attend_ker_dim<128><<<grid, threads, mem_size>>>(N, tma_q_d, tma_k_d, tma_v_d, tma_o_d);
    }
    
    CHECK_CUDA_ERROR(hipGetLastError());
}
#else
#include "harness_a100_fwd.impl"
#endif

